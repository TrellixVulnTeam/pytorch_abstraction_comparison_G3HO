//
// Created by Maksim Levental on 11/5/20.
//

#include <addition.cuh>

Tensor<float> *Addition::add(Tensor<float> *A, Tensor<float> *B) {
    fwd_initialize(A);
    // C = A + B
    checkCudnnErrors(hipdnnOpTensor(
        cuda_->cudnn(),
        op_descriptor,
        &cuda_->one,
        A->tensor_descriptor(),
        A->get_device_ptr(),
        &cuda_->one,
        B->tensor_descriptor(),
        B->get_device_ptr(),
        &cuda_->zero,
        output_->tensor_descriptor(),
        output_->get_device_ptr()));

    return output_;
}
Tensor<float> *Addition::forward(Tensor<float> *input) { exit(EXIT_FAILURE); }
Tensor<float> *Addition::backward(Tensor<float> *grad_input) { exit(EXIT_FAILURE); }
void Addition::fwd_initialize(Tensor<float> *A) {
    Layer::fwd_initialize(A);
    if (op_descriptor == nullptr) {
        checkCudnnErrors(hipdnnCreateOpTensorDescriptor(&op_descriptor));
        checkCudnnErrors(hipdnnSetOpTensorDescriptor(
            op_descriptor, HIPDNN_OP_TENSOR_ADD, HIPDNN_DATA_FLOAT, HIPDNN_PROPAGATE_NAN));
    }
}
Addition::~Addition() = default;
