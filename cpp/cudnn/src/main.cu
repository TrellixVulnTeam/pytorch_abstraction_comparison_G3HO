#include "hip/hip_runtime.h"
//#include "CLI11.hpp"
#include "datasets/datasets.h"
#include "network.h"
#include "resnet.cuh"
#include <cassert>
#include <cmath>

#include <hip/hip_runtime_api.h>
#include <iomanip>
#include <nvtx3/nvToolsExt.h>

template <typename dtype> int get_tp_count(Tensor<dtype> *output, Tensor<dtype> *target);
template <typename dtype> int arg_max(int batch, int output_size, const dtype *arr);
template <typename dtype> int find_one(int batch, int output_size, const dtype *arr);

template <typename dtype> void train() {
    int batch_size = 512;

    int epochs = 100;
    int monitoring_step = 20;

    double learning_rate = 0.001;
    double lr_decay = 0.0000005f;

    bool load_pretrain = false;
    bool file_save = false;

    std::cout << "== MNIST training with CUDNN ==" << std::endl;

    //    auto train_data_loader = MNIST<dtype>(
    //        "/home/maksim/dev_projects/pytorch_abstraction_comparison/data/MNIST/raw/"
    //        "train-images-idx3-ubyte",
    //        "/home/maksim/dev_projects/pytorch_abstraction_comparison/data/MNIST/raw/"
    //        "train-labels-idx1-ubyte",
    //        true,
    //        batch_size,
    //        NUMBER_MNIST_CLASSES);
    //    auto test_data_loader = MNIST<dtype>(
    //        "/home/maksim/dev_projects/pytorch_abstraction_comparison/data/MNIST/raw/"
    //        "t10k-images-idx3-ubyte",
    //        "/home/maksim/dev_projects/pytorch_abstraction_comparison/data/MNIST/raw/"
    //        "t10k-labels-idx1-ubyte",
    //        false,
    //        batch_size,
    //        NUMBER_MNIST_CLASSES);
    //    auto test_data_loader = STL10<dtype>(
    //        "/home/maksim/dev_projects/pytorch_abstraction_comparison/data/stl_10_train_data.npy",
    //        "/home/maksim/dev_projects/pytorch_abstraction_comparison/data/stl_10_train_labels.npy",
    //        true,
    //        batch_size,
    //        NUMBER_STL10_CLASSES);
    //    auto train_data_loader = STL10<dtype>(
    //        "/home/maksim/dev_projects/pytorch_abstraction_comparison/data/stl_10_test_data.npy",
    //        "/home/maksim/dev_projects/pytorch_abstraction_comparison/data/stl_10_test_labels.npy",
    //        false,
    //        batch_size,
    //        NUMBER_STL10_CLASSES);
    auto train_data_loader = CIFAR10<dtype>(
        "/home/maksim/dev_projects/pytorch_abstraction_comparison/data/cifar-10-batches-bin/"
        "all_train_data.bin",
        "",
        true,
        batch_size,
        NUMBER_CIFAR10_CLASSES);
    auto test_data_loader = CIFAR10<dtype>(
        "/home/maksim/dev_projects/pytorch_abstraction_comparison/data/cifar-10-batches-bin/"
        "test_batch.bin",
        "",
        false,
        batch_size,
        NUMBER_CIFAR10_CLASSES);

    CrossEntropyLoss<dtype> criterion;
    CrossEntropyLoss<dtype> criterion1;

    auto model = make_resnet50<dtype>();
    model->cuda();
    //    auto model = new Network<dtype>();
    //    model->add_layer(new Conv2d<dtype>("conv1", 20, 5));
    //    model->add_layer(new Activation<dtype>("relu1", HIPDNN_ACTIVATION_RELU));
    //    model->add_layer(new Pooling<dtype>("pool1", 2, 2, 0, HIPDNN_POOLING_MAX));
    //    model->add_layer(new Conv2d<dtype>("conv2", 50, 5));
    //    model->add_layer(new Activation<dtype>("relu2", HIPDNN_ACTIVATION_RELU));
    //    model->add_layer(new Pooling<dtype>("pool2", 2, 2, 0, HIPDNN_POOLING_MAX));
    //    model->add_layer(new Dense<dtype>("dense1", 500));
    //    model->add_layer(new Activation<dtype>("relu3", HIPDNN_ACTIVATION_RELU));
    //    model->add_layer(new Dense<dtype>("dense2", 10));
    //    model->add_layer(new Softmax<dtype>("softmax"));
    //    model->cuda();
    checkCudaErrors(hipDeviceSynchronize());

    if (load_pretrain)
        model->load_pretrain();

    hipProfilerStart();

    Tensor<dtype> *train_data, *train_target;
    Tensor<dtype> *test_data, *test_target;
    Tensor<dtype> *output;
    double loss, accuracy, running_loss;
    int tp_count, running_tp_count, sample_count;

    std::string nvtx_message;
    for (int epoch = 0; epoch < epochs; epoch++) {
        std::cout << "[TRAIN]" << std::endl;
        model->train();
        loss = accuracy = running_loss = 0;
        tp_count = running_tp_count = sample_count = 0;
        learning_rate = 0.1;
        train_data_loader.reset();

        for (int batch = 0; batch < train_data_loader.get_num_batches(); batch++) {
            nvtx_message =
                std::string("epoch " + std::to_string(epoch) + " batch " + std::to_string(batch));
            nvtxRangePushA(nvtx_message.c_str());

            std::tie(train_data, train_target) = train_data_loader.get_next_batch();
            train_data->to(cuda);
            train_target->to(cuda);

            output = model->forward(train_data);
            tp_count += get_tp_count<dtype>(output, train_target);
            loss += criterion.loss(output, train_target);
            sample_count += batch_size;

            model->backward(train_target);
            //            learning_rate *= 1.f / (1.f + lr_decay * batch);
            model->update(learning_rate);

            nvtxRangePop();

            if (batch % monitoring_step == 0) {
                //                train_data->print("data", true, batch_size);
                //                output->print("output", true, batch_size);
                //                train_target->print("target", true, batch_size);

                accuracy = 100.f * tp_count / sample_count;
                std::cout << "epoch: " << std::right << std::setw(4) << epoch
                          << ", batch: " << std::right << std::setw(4) << batch
                          << ", avg loss: " << std::left << std::setw(8) << std::fixed
                          << std::setprecision(6) << loss / (float)sample_count
                          << ", accuracy: " << accuracy << "%"
                          << ", lr: " << learning_rate;
                std::cout << std::endl;
                running_loss += loss;
                running_tp_count += tp_count;
                tp_count = 0;
                sample_count = 0;
                loss = 0;
            }
        }

        std::cout << "train avg loss: " << std::left << std::setw(8) << std::fixed
                  << std::setprecision(6) << running_loss / train_data_loader.len()
                  << ", accuracy: " << 100.f * running_tp_count / train_data_loader.len() << "%";
        std::cout << std::endl;
        tp_count = 0;
        loss = 0;

        if (file_save)
            model->write_file();

        std::cout << "[EVAL]" << std::endl;

        model->eval();
        test_data_loader.reset();

        for (int batch = 0; batch < test_data_loader.get_num_batches(); batch++) {
            std::string nvtx_message = std::string("batch " + std::to_string(batch));
            nvtxRangePushA(nvtx_message.c_str());

            std::tie(test_data, test_target) = test_data_loader.get_next_batch();
            test_data->to(cuda);
            test_target->to(cuda);

            output = model->forward(test_data);
            tp_count += get_tp_count<dtype>(output, test_target);
            sample_count += batch_size;
            loss += criterion1.loss(output, test_target);

            nvtxRangePop();
            if (batch % monitoring_step == 0) {
                //                test_data->print("data", true, batch_size);
                //                output->print("output", true, batch_size);
                //                test_target->print("target", true, batch_size);
            }
        }

        accuracy = 100.f * tp_count / test_data_loader.len();
        std::cout << "eval avg loss: " << std::setw(4) << loss / (float)test_data_loader.len()
                  << ", accuracy: " << accuracy << "%" << std::endl;
        std::cout << std::endl;
    }

    hipProfilerStop();
    std::cout << "Done." << std::endl;
}

int main(int argc, char *argv[]) {
    //    CLI::App app{"CUDNN Harness"};
    //
    //    std::string train_dataset_fp = "default";
    //    std::string train_label_fp = "default";
    //    app.add_option("--train_dataset_fp", train_dataset_fp, "dataset file path");
    //    app.add_option("--train_label_fp", train_label_fp, "label file path");
    //
    //    std::string test_dataset_fp = "default";
    //    std::string test_label_fp = "default";
    //    app.add_option("--test_dataset_fp", test_dataset_fp, "dataset file path");
    //    app.add_option("--test_label_fp", test_label_fp, "label file path");
    //
    //    CLI11_PARSE(app, argc, argv);

    /* configure the network */
    train<float>();

    return 0;
}

template <typename dtype> int get_tp_count(Tensor<dtype> *output, Tensor<dtype> *target) {
    int batch_size = output->get_batch_size();
    int output_size = output->size();

    assert(batch_size == target->get_batch_size());
    assert(output_size == target->size());

    dtype *h_output, *h_target;
    int idx_output, idx_target;
    int tp_count = 0;

    // get predicts and targets
    h_output = output->to(host);
    h_target = target->to(host);

    // idx_output = idx_target = 0;
    for (int b = 0; b < batch_size; b++) {
        idx_output = arg_max<dtype>(b, output_size, h_output);
        idx_target = find_one<dtype>(b, output_size, h_target);
        if (idx_output == idx_target)
            tp_count++;
    }

    return tp_count;
}

template <typename dtype> int arg_max(int batch, int output_size, const dtype *arr) {
    int idx_output = 0;
    for (int i = 1; i < NUMBER_MNIST_CLASSES; i++) {
        if (arr[batch * output_size + i] > arr[batch * output_size + idx_output])
            idx_output = i;
    }
    return idx_output;
}

template <typename dtype> int find_one(int batch, int output_size, const dtype *arr) {
    for (int i = 0; i < 10; i++) {
        if (abs(arr[batch * output_size + i] - 1) < 1e-10) {
            return i;
        }
    }
    exit(EXIT_FAILURE);
}
