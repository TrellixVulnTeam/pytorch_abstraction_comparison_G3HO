#include "hip/hip_runtime.h"
#include <layer.h>

#include <random>

#include <algorithm>
#include <cassert>
#include <cmath>

#include <fstream>
#include <iostream>
#include <sstream>


/****************************************************************
 * Layer definition                                             *
 ****************************************************************/
Layer::Layer() { /* do nothing */
}

Layer::~Layer() {
    if (DEBUG_FORWARD > 0 || DEBUG_BACKWARD > 0)
        std::cout << "Destroy Layer: " << name_ << std::endl;

    if (output_ != nullptr) {
        delete output_;
        output_ = nullptr;
    }
    if (grad_input_ != nullptr) {
        delete grad_input_;
        grad_input_ = nullptr;
    }

    if (weights_ != nullptr) {
        delete weights_;
        weights_ = nullptr;
    }
    if (biases_ != nullptr) {
        delete biases_;
        biases_ = nullptr;
    }
    if (grad_weights_ != nullptr) {
        delete grad_weights_;
        grad_weights_ = nullptr;
    }
    if (grad_biases_ != nullptr) {
        delete grad_biases_;
        grad_biases_ = nullptr;
    }
}

void Layer::init_weight_bias(unsigned int seed) {
    checkCudaErrors(hipDeviceSynchronize());

    if (weights_ == nullptr || biases_ == nullptr)
        return;
    PRINT("init weights biases");
    // Create random network
    std::random_device rd;
    std::mt19937 gen(seed == 0 ? rd() : static_cast<unsigned int>(seed));

    // He uniform distribution
    // TODO: initialization Xi
    float range = sqrt(6.f / input_size_); // He's initialization
    std::uniform_real_distribution<> dis(-range, range);

    for (int i = 0; i < weights_->len(); i++)
        weights_->get_host_ptr()[i] = static_cast<float>(dis(gen));
    for (int i = 0; i < biases_->len(); i++)
        biases_->get_host_ptr()[i] = 0.f;

    // copy initialized value to the device
    weights_->to(DeviceType::cuda);
    biases_->to(DeviceType::cuda);

    std::cout << ".. initialized " << name_ << " layer .." << std::endl;
}

void Layer::update_weights_biases(float learning_rate) {
    float eps = -1.f * learning_rate;
    if (weights_ != nullptr && grad_weights_ != nullptr) {
        if (DEBUG_UPDATE) {
            weights_->print(name_ + "::weights (before update)", true);
            grad_weights_->print(name_ + "::gweights", true);
        }

        // w = w + eps * dw
        checkCublasErrors(hipblasSaxpy(
            cuda_->cublas(),
            weights_->len(),
            &eps,
            grad_weights_->get_device_ptr(),
            1,
            weights_->get_device_ptr(),
            1));

        if (DEBUG_UPDATE)
            weights_->print(name_ + "weights (after update)", true);
    }

    if (biases_ != nullptr && grad_biases_ != nullptr) {
        if (DEBUG_UPDATE) {
            biases_->print(name_ + "biases (before update)", true);
            grad_biases_->print(name_ + "gbiases", true);
        }

        // b = b + eps * db
        checkCublasErrors(hipblasSaxpy(
            cuda_->cublas(),
            biases_->len(),
            &eps,
            grad_biases_->get_device_ptr(),
            1,
            biases_->get_device_ptr(),
            1));

        if (DEBUG_UPDATE)
            biases_->print(name_ + "biases (after update)", true);
    }
}

void Layer::fwd_initialize(Tensor<float> *input) {
    if (input_desc_ == nullptr || batch_size_ != input->get_batch_size()) {
        //        input_ = input;
        input_size_ = input->size();
        input_desc_ = input->tensor_descriptor();
        batch_size_ = input->get_batch_size();

        if (output_ == nullptr)
            output_ = new Tensor<float>(input->shape());
        else
            output_->reset(input->shape());

        output_desc_ = output_->tensor_descriptor();
    }
}

void Layer::bwd_initialize(Tensor<float> *grad_output) {
    if (grad_input_ == nullptr || batch_size_ != grad_output->get_batch_size()) {
        grad_output_ = grad_output;

        if (grad_input_ == nullptr)
            grad_input_ = new Tensor<float>(input_->shape());
        else
            grad_input_->reset(input_->shape());
    }
}

int Layer::load_parameter() {
    std::stringstream filename_weights, filename_biases;

    // load weights and biases pretrained parameters
    filename_weights << name_ << ".bin";
    if (weights_->file_read(filename_weights.str()))
        return -1;

    filename_biases << name_ << ".bias.bin";
    if (biases_->file_read(filename_biases.str()))
        return -2;

    std::cout << ".. loaded " << name_ << " pretrain parameter.." << std::endl;

    return 0;
}

int Layer::save_parameter() {
    std::stringstream filename_weights, filename_biases;

    std::cout << ".. saving " << name_ << " parameter ..";

    // Write weights file
    if (weights_) {
        filename_weights << name_ << ".bin";
        if (weights_->file_write(filename_weights.str()))
            return -1;
    }

    // Write bias file
    if (biases_) {
        filename_biases << name_ << ".bias.bin";
        if (biases_->file_write(filename_biases.str()))
            return -2;
    }

    std::cout << " done .." << std::endl;

    return 0;
}
