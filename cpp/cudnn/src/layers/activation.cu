//
// Created by Maksim Levental on 10/29/20.
//

#include <layers/activation.cuh>

/****************************************************************
 * Activation Layer                                             *
 ****************************************************************/

Activation::Activation(std::string name, hipdnnActivationMode_t mode, double coef) {
    name_ = std::move(name);
    act_mode_ = mode;
    act_coef_ = coef;

    hipdnnCreateActivationDescriptor(&act_desc_);
    hipdnnSetActivationDescriptor(act_desc_, act_mode_, HIPDNN_PROPAGATE_NAN, act_coef_);
}

Activation::~Activation() { hipdnnDestroyActivationDescriptor(act_desc_); }

Tensor<double> *Activation::forward(Tensor<double> *input) {
    fwd_initialize(input);
    input_ = input;
    checkCudnnErrors(hipdnnActivationForward(
        cuda_->cudnn(),
        act_desc_,
        &cuda_->one,
        input_desc_,
        input->get_device_ptr(),
        &cuda_->zero,
        output_desc_,
        output_->get_device_ptr()));

    return output_;
}

Tensor<double> *Activation::backward(Tensor<double> *grad_of_output) {
    bwd_initialize(grad_of_output);
    checkCudnnErrors(hipdnnActivationBackward(
        cuda_->cudnn(),
        act_desc_,
        &cuda_->one,
        output_desc_,
        output_->get_device_ptr(),
        output_desc_,
        grad_of_output->get_device_ptr(),
        input_desc_,
        input_->get_device_ptr(),
        &cuda_->zero,
        input_desc_,
        grad_of_input_->get_device_ptr()));

    return grad_of_input_;
}
