//
// Created by Maksim Levental on 10/29/20.
//

#include <layers/pooling.cuh>

Pooling::Pooling(
    std::string name, int kernel_size, int stride, int padding, hipdnnPoolingMode_t mode)
    : kernel_size_(kernel_size), padding_(padding), stride_(stride), mode_(mode) {
    name_ = std::move(name);

    hipdnnCreatePoolingDescriptor(&pool_desc_);
    hipdnnSetPooling2dDescriptor(
        pool_desc_,
        mode_,
        HIPDNN_PROPAGATE_NAN,
        kernel_size_,
        kernel_size_,
        padding_,
        padding_,
        stride_,
        stride_);
}

Pooling::~Pooling() { hipdnnDestroyPoolingDescriptor(pool_desc_); }

void Pooling::fwd_initialize(Tensor<double> *input) {
    if (input_desc_ == nullptr || batch_size_ != input->get_batch_size()) {
        input_size_ = input->size();
        // resource initialize
        input_desc_ = input->tensor_descriptor();
        batch_size_ = input->get_batch_size();

        // setting output
        hipdnnGetPooling2dForwardOutputDim(
            pool_desc_,
            input_desc_,
            &output_size_[0],
            &output_size_[1],
            &output_size_[2],
            &output_size_[3]);
        if (output_ == nullptr)
            output_ = new Tensor<double>(output_size_);
        else
            output_->reset(output_size_);

        output_desc_ = output_->tensor_descriptor();
    }
}

Tensor<double> *Pooling::forward(Tensor<double> *input) {
    fwd_initialize(input);
    input_ = input;
    hipdnnPoolingForward(
        cuda_->cudnn(),
        pool_desc_,
        &cuda_->one,
        input_desc_,
        input->get_device_ptr(),
        &cuda_->zero,
        output_desc_,
        output_->get_device_ptr());

    return output_;
}

Tensor<double> *Pooling::backward(Tensor<double> *grad_of_output) {
    bwd_initialize(grad_of_output);
    checkCudnnErrors(hipdnnPoolingBackward(
        cuda_->cudnn(),
        pool_desc_,
        &cuda_->one,
        output_desc_,
        output_->get_device_ptr(),
        output_desc_,
        grad_of_output->get_device_ptr(),
        input_desc_,
        input_->get_device_ptr(),
        &cuda_->zero,
        input_desc_,
        grad_of_input_->get_device_ptr()));

    return grad_of_input_;
}
