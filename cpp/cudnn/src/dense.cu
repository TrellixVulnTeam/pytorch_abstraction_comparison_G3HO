#include "hip/hip_runtime.h"
//
// Created by Maksim Levental on 10/29/20.
//

#include <dense.cuh>

/****************************************************************
 * Dense Layer                                                  *
 ****************************************************************/

Dense::Dense(std::string name, int output_size) {
    name_ = std::move(name);
    output_size_ = output_size;
}

Dense::~Dense() {
    if (d_one_vec != nullptr) {
        hipFree(d_one_vec);
        d_one_vec = nullptr;
    }
}

__global__ void init_one_vec(float *d_one_vec, size_t length) {
    int i = blockIdx.x * blockDim.x + threadIdx.x;

    if (i >= length)
        return;

    d_one_vec[i] = 1.f;
}

void Dense::fwd_initialize(Tensor<float> *input) {
    // initialize weights and biases
    if (weights_ == nullptr) {
        // setup parameter size information
        input_size_ = input->get_channels() * input->get_height() * input->get_height();

        // initialize weight, bias, and output
        weights_ = new Tensor<float>(1, 1, input_size_, output_size_);
        biases_ = new Tensor<float>(1, 1, output_size_);
    }

    // initilaize input and output
    if (input_ == nullptr || batch_size_ != input->get_batch_size()) {
        input_ = input;
        batch_size_ = input->get_batch_size();

        if (output_ == nullptr)
            output_ = new Tensor<float>(batch_size_, output_size_);
        else
            output_->reset(batch_size_, output_size_);

        output_->tensor_descriptor();

        if (d_one_vec != nullptr)
            hipFree(d_one_vec);
        checkCudaErrors(hipMalloc((void **)&d_one_vec, sizeof(float) * batch_size_));
        init_one_vec<<<(batch_size_ + BLOCK_DIM_1D - 1) / BLOCK_DIM_1D, BLOCK_DIM_1D>>>(
            d_one_vec, batch_size_);

        // initialize weights and biases
        if (load_pretrain_ && !freeze_) {
            if (load_parameter()) {
                std::cout << "error occurred.." << std::endl;
                exit(-1);
            }
        } else if (!freeze_) {
            init_weight_bias();
        } else {
            /* do nothing */
        }
    }
}

Tensor<float> *Dense::forward(Tensor<float> *input) {
    // output = weights^T * input (without biases)
    checkCublasErrors(hipblasSgemm(
        cuda_->cublas(),
        HIPBLAS_OP_T,
        HIPBLAS_OP_N,
        output_size_,
        batch_size_,
        input_size_,
        &cuda_->one,
        weights_->get_device_ptr(),
        input_size_,
        input_->get_device_ptr(),
        input_size_,
        &cuda_->zero,
        output_->get_device_ptr(),
        output_size_));

    // output += biases * d_one_vec^T
    checkCublasErrors(hipblasSgemm(
        cuda_->cublas(),
        HIPBLAS_OP_N,
        HIPBLAS_OP_N,
        output_size_,
        batch_size_,
        1,
        &cuda_->one,
        biases_->get_device_ptr(),
        output_size_,
        d_one_vec,
        1,
        &cuda_->one,
        output_->get_device_ptr(),
        output_size_));

#if (DEBUG_DENSE & 0x01)
    input_->print(name_ + "::input", true);
    weights_->print(name_ + "::weight", true);
    biases_->print(name_ + "::bias", true);
    output_->print(name_ + "::output", true);
#endif // DEBUG_DENSE

    return output_;
}

void Dense::bwd_initialize(Tensor<float> *grad_output) {
    if (grad_weights_ == nullptr) {
        grad_weights_ = new Tensor<float>(weights_->shape());
        grad_biases_ = new Tensor<float>(biases_->shape());
    }
    Layer::bwd_initialize(grad_output);
}

Tensor<float> *Dense::backward(Tensor<float> *grad_output) {
    // db = (dy) * d_one_vec
    hipblasSgemv(
        cuda_->cublas(),
        HIPBLAS_OP_N,
        output_size_,
        batch_size_,
        &cuda_->one,
        grad_output_->get_device_ptr(),
        output_size_,
        d_one_vec,
        1,
        &cuda_->zero,
        grad_biases_->get_device_ptr(),
        1);

    // dw = x * (dy)^T
    hipblasSgemm(
        cuda_->cublas(),
        HIPBLAS_OP_N,
        HIPBLAS_OP_T,
        input_size_,
        output_size_,
        batch_size_,
        &cuda_->one,
        input_->get_device_ptr(),
        input_size_,
        grad_output_->get_device_ptr(),
        output_size_,
        &cuda_->zero,
        grad_weights_->get_device_ptr(),
        input_size_);

    // dx = W * dy
    if (!gradient_stop_)
        hipblasSgemm(
            cuda_->cublas(),
            HIPBLAS_OP_N,
            HIPBLAS_OP_N,
            input_size_,
            batch_size_,
            output_size_,
            &cuda_->one,
            weights_->get_device_ptr(),
            input_size_,
            grad_output_->get_device_ptr(),
            output_size_,
            &cuda_->zero,
            grad_input_->get_device_ptr(),
            input_size_);

#if (DEBUG_DENSE & 0x02)
    std::cout << name_ << "[BACKWARD]" << std::endl;
    grad_output->print(name_ + "::gradients", true, grad_output->n());
    grad_weights_->print(name_ + "::gfilter", true);
    grad_biases_->print(name_ + "::gbias", true);
    if (!gradient_stop_)
        grad_input_->print(name_ + "::gdata", true);
#endif // DEBUG_DENSE

    return grad_input_;
}
